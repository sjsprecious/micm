
#include <hip/hip_runtime.h>
#include <iostream>

namespace micm
{
  namespace cuda
  {
    // flipped memory layout
    __global__ void AddForcingTerms_kernel(
        double* rate_constants,
        double* state_variables, 
        double* forcing,
        int ngrids,
        int nrxns,
        int nspecs,
        size_t* number_of_reactants_,
        size_t* reactant_ids_,
        size_t* number_of_products_,
        size_t* product_ids_,
        double* yields_)
    {
      // define thread index
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      size_t react_id_offset, prod_id_offset, yield_offset;

      if (tid < ngrids)
      {
        react_id_offset = 0;
        prod_id_offset = 0;
        yield_offset = 0;
        for (std::size_t i_rxn = 0; i_rxn < nrxns; ++i_rxn)
        {
          double rate = rate_constants[i_rxn * ngrids + tid];
          for (std::size_t i_react = 0; i_react < number_of_reactants_[i_rxn]; ++i_react)
            rate *= state_variables[reactant_ids_[react_id_offset + i_react] * ngrids + tid];
          for (std::size_t i_react = 0; i_react < number_of_reactants_[i_rxn]; ++i_react)
          {
            forcing[reactant_ids_[react_id_offset + i_react] * ngrids + tid] -= rate;
          }
          for (std::size_t i_prod = 0; i_prod < number_of_products_[i_rxn]; ++i_prod)
          {
            size_t index = product_ids_[prod_id_offset + i_prod] * ngrids + tid;
            forcing[index] += yields_[yield_offset + i_prod] * rate;
          }
          react_id_offset += number_of_reactants_[i_rxn];
          prod_id_offset += number_of_products_[i_rxn];
          yield_offset += number_of_products_[i_rxn];
        }  // for loop over number of reactions
      }    // if check for valid CUDA threads
    }      // end of AddForcingTerms_kernel


  __global__ void AddJacobianTerms_kernel(
    double* rate_constants,
    double* state_variables,
    size_t n_grids,
    size_t n_reactions,
    double* jacobian,
    size_t* number_of_reactants,
    size_t* reactant_ids,
    size_t* number_of_products,
    size_t* product_ids,
    double* yields,
    size_t* jacobian_flat_ids){
    
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t react_ids_offset = 0;
    size_t yields_offset = 0; 
    size_t flat_id_offset = 0; 
    if (tid < n_grids){
      printf("tid: %d\n", tid); 
    //loop over reactions in a grid
    for (size_t i_rxn = 0; i_rxn < n_reactions; ++i_rxn){
       //loop over reactants in a reaction
      for (size_t i_ind = 0; i_ind < number_of_reactants[i_rxn]; ++i_ind){
        double d_rate_d_ind = rate_constants[i_rxn * n_grids + tid]; 
        printf("d_rate_d_ind before modification: %g\n", d_rate_d_ind); 
        for(size_t i_react = 0; i_react < number_of_reactants[i_rxn]; ++i_react){
          if(i_react != i_ind){
            printf("state variable index: %d\n", reactant_ids[react_ids_offset + i_react] * n_grids + tid);
            printf("state variable value: %g\n", state_variables[reactant_ids[react_ids_offset + i_react] * n_grids + tid]); 
            d_rate_d_ind *= state_variables[reactant_ids[react_ids_offset + i_react] * n_grids + tid]; 
          }
        }
        for(size_t i_dep = 0; i_dep < number_of_reactants[i_rxn]; ++i_dep){
          
          size_t jacobian_idx = jacobian_flat_ids[flat_id_offset] + tid; 
          printf("flat id: %d\n",jacobian_flat_ids[flat_id_offset] ); 
          printf("jaocbian index %d\n",jacobian_idx );
          printf("d_rate_d_ind: %g\n", d_rate_d_ind);
          printf("jacobian value before subtraction: %g\n", jacobian[jacobian_idx]);
          jacobian[jacobian_idx] -= d_rate_d_ind; 
          
          printf("jacobian value after subtraction: %g\n", jacobian[jacobian_idx]);
          flat_id_offset++; 
        }
        for(size_t i_dep = 0; i_dep < number_of_products[i_rxn]; ++i_dep){
          size_t jacobian_idx = jacobian_flat_ids[flat_id_offset] + tid; 
          jacobian[jacobian_idx] += yields[yields_offset + i_dep] * d_rate_d_ind; 
          flat_id_offset++;
        }
        react_ids_offset += number_of_reactants[i_rxn]; 
        yields_offset += number_of_products[i_rxn]; 
      }//loop over reactants in a reaction
    }//loop over reactions in a grid
    }//check valid tid 
  }// end of AddJacobianTerms_kernel
    
    void AddJacobianTerms_kernelSetup(
        const double* rate_constants, 
        const double* state_variables, 
        size_t n_grids, 
        size_t n_reactions, 
        size_t n_species,
        double* jacobian, 
        size_t jacobian_size, 
        const size_t* number_of_reactants, 
        const size_t* reactant_ids, 
        size_t reactant_ids_size, 
        const size_t* number_of_products, 
        const size_t* product_ids, 
        size_t product_ids_size, 
        const double* yields, 
        size_t yields_size, 
        const size_t* jacobian_flat_ids, 
        size_t jacobian_flat_ids_size){
        
        //create device pointers 
        double* d_rate_constants;
        double* d_state_variables; 
        double* d_jacobian;
        size_t* d_number_of_reactants; 
        size_t* d_reactant_ids;
        size_t* d_number_of_products;
        size_t* d_product_ids;
        double* d_yields;
        size_t* d_jacobian_flat_ids;

        //allocate device memory 
        hipMalloc(&d_rate_constants, sizeof(double)* n_grids*n_reactions); 
        hipMalloc(&d_state_variables, sizeof(double)* n_grids*n_species); 
        hipMalloc(&d_jacobian, sizeof(double)* jacobian_size); 
        hipMalloc(&d_number_of_reactants, sizeof(size_t)* n_reactions); 
        hipMalloc(&d_reactant_ids, sizeof(size_t)* reactant_ids_size); 
        hipMalloc(&d_number_of_products, sizeof(size_t)* n_reactions); 
        hipMalloc(&d_product_ids, sizeof(size_t)* product_ids_size); 
        hipMalloc(&d_yields, sizeof(double)* yields_size); 
        hipMalloc(&d_jacobian_flat_ids, sizeof(size_t)* jacobian_flat_ids_size); 

        //transfer data from host to device 
        hipMemcpy(d_rate_constants, rate_constants, sizeof(double)* n_grids*n_reactions, hipMemcpyHostToDevice); 
        hipMemcpy(d_state_variables, state_variables, sizeof(double)* n_grids*n_species, hipMemcpyHostToDevice); 
        hipMemcpy(d_jacobian, jacobian, sizeof(double)* jacobian_size, hipMemcpyHostToDevice); 
        hipMemcpy(d_number_of_reactants, number_of_reactants, sizeof(size_t)* n_reactions, hipMemcpyHostToDevice); 
        hipMemcpy(d_reactant_ids, reactant_ids, sizeof(size_t)* reactant_ids_size, hipMemcpyHostToDevice);
        hipMemcpy(d_number_of_products, number_of_products, sizeof(size_t)* n_reactions, hipMemcpyHostToDevice); 
        hipMemcpy(d_product_ids, product_ids, sizeof(size_t) * product_ids_size, hipMemcpyHostToDevice); 
        hipMemcpy(d_yields, yields, sizeof(double)* yields_size, hipMemcpyHostToDevice); 
        hipMemcpy(d_jacobian_flat_ids, jacobian_flat_ids, sizeof(size_t)* jacobian_flat_ids_size, hipMemcpyHostToDevice); 

        //setup kernel
        size_t threads_per_block = 320; 
        size_t total_blocks = (n_grids + threads_per_block -1)/threads_per_block; 
        //launch kernel
        AddJacobianTerms_kernel<<<total_blocks, threads_per_block>>>(
            d_rate_constants,
            d_state_variables,
            n_grids, 
            n_reactions, 
            d_jacobian,
            d_number_of_reactants, 
            d_reactant_ids,
            d_number_of_products,
            d_product_ids,
            d_yields, 
            d_jacobian_flat_ids); 
        hipDeviceSynchronize(); 
        hipMemcpy(jacobian, d_jacobian, sizeof(double)* jacobian_size, hipMemcpyDeviceToHost); 
        //clean up
        hipFree(d_rate_constants); 
        hipFree(d_state_variables); 
        hipFree(d_jacobian); 
        hipFree(d_number_of_reactants); 
        hipFree(d_reactant_ids); 
        hipFree(d_number_of_products); 
        hipFree(d_product_ids); 
        hipFree(d_yields); 
        hipFree(d_jacobian_flat_ids); 
    } //end of AddJacobian_kernelSetup
    
    void AddForcingTerms_kernelSetup(
        const double* rate_constants_data,
        const double* state_variables_data,
        double* forcing_data,
        int ngrids,
        int nrxns,
        int nspecs,
        const size_t* number_of_reactants,
        int number_of_reactants_size,
        const size_t* reactant_ids,
        int reactant_ids_size,
        const size_t* number_of_products,
        int number_of_products_size,
        const size_t* product_ids,
        int product_ids_size,
        const double* yields,
        int yields_size)
    {
      // device pointer to vectorss
      double* d_rate_constants;
      double* d_state_variables;
      double* d_forcing;
      double* d_yields_;
      size_t* d_number_of_reactants_;
      size_t* d_reactant_ids_;
      size_t* d_number_of_products_;
      size_t* d_product_ids_;

      // allocate device memory
      size_t rate_constants_bytes = sizeof(double) * (ngrids * nrxns);
      size_t state_forcing_bytes = sizeof(double) * (ngrids * nspecs);
      size_t yields_bytes = sizeof(double) * yields_size;
      size_t number_of_reactants_bytes = sizeof(size_t) * number_of_reactants_size;
      size_t reactant_ids_bytes = sizeof(size_t) * reactant_ids_size;
      size_t number_of_products_bytes = sizeof(size_t) * number_of_products_size;
      size_t product_ids_bytes = sizeof(size_t) * product_ids_size;

      hipMalloc(&d_rate_constants, rate_constants_bytes);
      hipMalloc(&d_state_variables, state_forcing_bytes);
      hipMalloc(&d_forcing, state_forcing_bytes);
      hipMalloc(&d_number_of_reactants_, number_of_reactants_bytes);
      hipMalloc(&d_reactant_ids_, reactant_ids_bytes);
      hipMalloc(&d_number_of_products_, number_of_products_bytes);
      hipMalloc(&d_product_ids_, product_ids_bytes);
      hipMalloc(&d_yields_, yields_bytes);

      // copy data from host memory to device memory
      hipMemcpy(d_rate_constants, rate_constants_data, rate_constants_bytes, hipMemcpyHostToDevice);
      hipMemcpy(d_state_variables, state_variables_data, state_forcing_bytes, hipMemcpyHostToDevice);
      hipMemcpy(d_forcing, forcing_data, state_forcing_bytes, hipMemcpyHostToDevice);
      hipMemcpy(d_number_of_reactants_, number_of_reactants, number_of_reactants_bytes, hipMemcpyHostToDevice);
      hipMemcpy(d_reactant_ids_, reactant_ids, reactant_ids_bytes, hipMemcpyHostToDevice);
      hipMemcpy(d_number_of_products_, number_of_products, number_of_products_bytes, hipMemcpyHostToDevice);
      hipMemcpy(d_product_ids_, product_ids, product_ids_bytes, hipMemcpyHostToDevice);
      hipMemcpy(d_yields_, yields, yields_bytes, hipMemcpyHostToDevice);

      // total thread count == number of grid cells
      int block_size = 32;
      int num_block = (ngrids + block_size - 1) / block_size;

      // kernel function call
      AddForcingTerms_kernel<<<num_block, block_size>>>(
          d_rate_constants,
          d_state_variables,
          d_forcing,
          ngrids,
          nrxns,
          nspecs,
          d_number_of_reactants_,
          d_reactant_ids_,
          d_number_of_products_,
          d_product_ids_,
          d_yields_);
      hipDeviceSynchronize();

      // copy data from device memory to host memory
      hipMemcpy(forcing_data, d_forcing, state_forcing_bytes, hipMemcpyDeviceToHost);

      // clean up
      hipFree(d_rate_constants);
      hipFree(d_state_variables);
      hipFree(d_forcing);
      hipFree(d_number_of_reactants_);
      hipFree(d_reactant_ids_);
      hipFree(d_number_of_products_);
      hipFree(d_product_ids_);
      hipFree(d_yields_);
    }  // end of AddForcingTerms_kernelSetup
  }    // namespace cuda
}  // namespace micm
